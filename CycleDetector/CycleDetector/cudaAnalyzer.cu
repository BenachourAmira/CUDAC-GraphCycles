#include "hip/hip_runtime.h"
#pragma once
#include"cudaAnalyzer.h"
#include "hip/hip_runtime.h"
#include ""
#include "vector"
#include "list"
#include <iostream>

struct cycles_t {
	int* paths;
	int* pathSizes;
	int pathsCount;
};

__global__ void searchFrom();

std::list<std::vector<int>> findCycles(int* matrix, config_t config) {
	std::list<std::vector<int>> cycles;

	dim3 block(8, 8, 8);
	dim3 grid(16,16);

	searchFrom<<<block, grid>>>();

	return cycles;
}

__global__ void searchFrom() {
	int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
	int threadPosInBlock = threadIdx.x +
		blockDim.x * threadIdx.y +
		blockDim.x * blockDim.y * threadIdx.z;
	int blockPosInGrid = blockIdx.x +
		gridDim.x * blockDim.y +
		gridDim.x * gridDim.y * blockIdx.z;
	int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;

	// TODO
}